#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *d_va, float *d_vb, float *d_x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  d_x[tid] = d_va[tid] + d_vb[tid];
}


//
// main code
//

int main(int argc, const char **argv)
{
  int   nblocks, nthreads, nsize, n; 
  float h_va[] = {0.0, 1.0, 10.0};
  float h_vb[] = {1.0, 0.0, 10.0};
  float *h_x;
  float *d_va;
  float *d_vb;
  float *d_x;

  // initialise card

  findCudaDevice(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 1;
  nthreads = sizeof(h_va)/ sizeof(float);
  nsize    = nblocks*nthreads ;

  // allocate memory for array
  h_x = (float *)malloc(sizeof(h_va)); 
 
  checkCudaErrors(hipMalloc((void **)&d_va, sizeof(h_va)));
  checkCudaErrors(hipMalloc((void **)&d_vb, sizeof(h_vb)));

  checkCudaErrors(hipMalloc((void **)&d_x, sizeof(h_va)));
 
  // move from host to device
  checkCudaErrors( hipMemcpy(d_va,h_va, sizeof(h_va),
                 hipMemcpyHostToDevice) );
  checkCudaErrors( hipMemcpy(d_vb,h_vb, sizeof(h_vb),
                 hipMemcpyHostToDevice) );

  // execute kernel
  
  my_first_kernel<<<nblocks,nthreads>>>(d_va, d_va, d_x);
  getLastCudaError("my_first_kernel execution failed\n");

  // copy back results and print them out

  checkCudaErrors( hipMemcpy(h_x,d_x, sizeof(h_x),
                 hipMemcpyDeviceToHost) );

  for (n=0; n<sizeof(h_va); n++) printf(" res %d= %f \n", n,h_x[n]);

  // free memory 

  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_va));
  checkCudaErrors(hipFree(d_vb));
  free(h_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
