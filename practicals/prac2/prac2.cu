#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float d_a, d_b, d_c;


////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////


__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, z, y1;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + N*blockIdx.x*blockDim.x;

  // path calculation

  s1 = 0.0f;

  for (int n=0; n<N; n++) {
    z = d_z[ind];
    y1 = d_a*z*z + d_b*z + d_c;
    s1 += y1;

    // version 1
    ind += blockDim.x;      // shift pointer to next element
  }

  // put av value into device array
  
  d_v[threadIdx.x + blockIdx.x*blockDim.x] = s1/N;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int     NPATH=9600000, h_N=100;
  float  *h_v, *d_v, *d_z;
  double  sum1;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*h_N*NPATH) );

  // define constants and transfer to GPU

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  
  float h_a = 10.0f;
  float h_b = 100.0f;
  float h_c = 1.0f;
  
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_a), &h_a, sizeof(h_a)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_b), &h_b, sizeof(h_b)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_c), &h_c, sizeof(h_c)) );
  
  // random number generation

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );

  hipEventRecord(start);
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, h_N*NPATH, 0.0f, 1.0f) );
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, h_N*NPATH/(0.001*milli));

  // execute kernel and time it

  hipEventRecord(start);
  pathcalc<<<NPATH/128, 128>>>(d_z, d_v);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  getLastCudaError("pathcalc execution failed\n");
  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f\n\n",
	 sum1/NPATH);

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}
