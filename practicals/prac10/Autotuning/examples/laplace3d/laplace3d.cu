#include "hip/hip_runtime.h"
//
// Program to solve Laplace equation on a regular 3D grid
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// define kernel block size
////////////////////////////////////////////////////////////////////////

// #define BLOCK_X 16       These are now defined by the autotuner
// #define BLOCK_Y 16

////////////////////////////////////////////////////////////////////////
// kernel function
////////////////////////////////////////////////////////////////////////

// Note: one thread per node in the 2D block;
// after initialisation it marches in the k-direction

__global__ void GPU_laplace3d(int NX, int NY, int NZ,
                              const float* __restrict__ d_u1,
                                    float* __restrict__ d_u2)
{
  int       i, j, k, IOFF, JOFF, KOFF;
  long long indg;
  float     u2, sixth=1.0f/6.0f;

  //
  // define global indices and array offsets
  //

  i    = threadIdx.x + blockIdx.x*BLOCK_X;
  j    = threadIdx.y + blockIdx.y*BLOCK_Y;
  indg = i + j*NX;

  IOFF = 1;
  JOFF = NX;
  KOFF = NX*NY;

  if ( i>=0 && i<=NX-1 && j>=0 && j<=NY-1 ) {

    for (k=0; k<NZ; k++) {

      if (i==0 || i==NX-1 || j==0 || j==NY-1 || k==0 || k==NZ-1) {
        u2 = d_u1[indg];  // Dirichlet b.c.'s
      }
      else {
        u2 = ( d_u1[indg-IOFF] + d_u1[indg+IOFF]
             + d_u1[indg-JOFF] + d_u1[indg+JOFF]
             + d_u1[indg-KOFF] + d_u1[indg+KOFF] ) * sixth;
      }
      d_u2[indg] = u2;

      indg += KOFF;
    }
  }
}

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

  int       NX=512, NY=512, NZ=512,
            REPEAT=200, bx, by, i, j, k;
  float    *h_u1, *h_u2,
           *d_u1, *d_u2, *d_foo;
  
  size_t    ind, bytes = sizeof(float) * NX*NY*NZ;

  printf("Grid dimensions: %d x %d x %d \n\n", NX, NY, NZ);

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory for arrays

  h_u1 = (float *)malloc(bytes);
  h_u2 = (float *)malloc(bytes);
  checkCudaErrors( hipMalloc((void **)&d_u1, bytes) );
  checkCudaErrors( hipMalloc((void **)&d_u2, bytes) );

  // initialise u1

  for (k=0; k<NZ; k++) {
    for (j=0; j<NY; j++) {
      for (i=0; i<NX; i++) {
        ind = i + j*NX + k*NX*NY;

        if (i==0 || i==NX-1 || j==0 || j==NY-1|| k==0 || k==NZ-1)
          h_u1[ind] = 1.0f;           // Dirichlet b.c.'s
        else
          h_u1[ind] = 0.0f;
      }
    }
  }

  // copy u1 to device

  hipEventRecord(start);
  checkCudaErrors( hipMemcpy(d_u1, h_u1, bytes,
                              hipMemcpyHostToDevice) );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("Copy u1 to device: %.1f (ms) \n\n", milli);

  // Set up the execution configuration

  bx = 1 + (NX-1)/BLOCK_X;
  by = 1 + (NY-1)/BLOCK_Y;

  dim3 dimGrid(bx,by);
  dim3 dimBlock(BLOCK_X,BLOCK_Y);

  // Execute GPU kernel

  hipEventRecord(start);

  for (i=0; i<REPEAT; i++) {
    GPU_laplace3d<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_u1, d_u2);
    getLastCudaError("GPU_laplace3d execution failed\n");

    d_foo = d_u1; d_u1 = d_u2; d_u2 = d_foo;   // swap d_u1 and d_u2
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("%dx GPU_laplace3d: %.1f (ms) \n\n", REPEAT, milli);

  // Read back GPU results

  hipEventRecord(start);
  checkCudaErrors( hipMemcpy(h_u2, d_u1, bytes, hipMemcpyDeviceToHost) );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("Copy u2 to host: %.1f (ms) \n\n", milli);

 // Release GPU and CPU memory

  checkCudaErrors( hipFree(d_u1) );
  checkCudaErrors( hipFree(d_u2) );
  free(h_u1);
  free(h_u2);

  hipDeviceReset();
}
