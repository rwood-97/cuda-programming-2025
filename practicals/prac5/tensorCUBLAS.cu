#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

// Function to check for errors.
void checkError(hipblasStatus_t status){
    if (status != HIPBLAS_STATUS_SUCCESS){
        printf("CUBLAS error\n");
        exit(EXIT_FAILURE);
    }
}

int main(void){

    // Variable to hold the status returned by cuBLAS.
    hipblasStatus_t status;

    // Variable that serves as a handle to the cuBLAS lib context.
    // This describes the environment in which our cuBLAS routines
    // are executed. It is initialised with cublasCreate().
    hipblasHandle_t handle;

    // Pointers to type float for input and output matrices.
    float *d_A, *d_B, *d_C;

    // Size of row or col in matrix (we will work with a square matrix).
    int n = 1024;

    // To get an accurate timing measurement we will run the same code
    // multiple times. Five times should be more than enough.
    int num_reps = 5;

    // We will execute the sgemm operation.
    // C = alpha A * B + beta C
    // alpha, beta are scalar, A, B, C are matrices.
    float alpha = 1.0f;
    float beta = 0.0f;

    // Declare a variable to hold out timings.
    float milliseconds = 0.0f;

    // Declare two cuda events for timing.
    hipEvent_t start, stop;


    // Allocate memory for the input and output matrices on the host.
    float *h_A = (float*)malloc(n * n * sizeof(float));
    float *h_B = (float*)malloc(n * n * sizeof(float));
    float *h_C = (float*)malloc(n * n * sizeof(float));

    // Initialise the matrices with random data.
    for (int i = 0; i < n * n; i++) {
        float A = (float)(rand() % 100);
        float B = (float)(rand() % 100);
        h_A[i] = A;
        h_B[i] = B;
    }

    // Allocate memory on the device for input and output matrices.
    hipMalloc((void**)&d_A, n * n * sizeof(float));
    hipMalloc((void**)&d_B, n * n * sizeof(float));
    hipMalloc((void**)&d_C, n * n * sizeof(float));

    // Copy the input data to the device
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // Create a cuBLAS handle
    status = hipblasCreate(&handle);
    checkError(status);

    // Create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Perform the matrix multiplication with Tensor Cores disabled.
    // Here we are using CUBLAS_PEDANTIC_MATH to disable the tensor cores on Volta.
    status = hipblasSetMathMode(handle, HIPBLAS_PEDANTIC_MATH);
    checkError(status);

    // Run the same code num_reps times to get a representative timing.
    // Please read  https://docs.nvidia.com/cuda/cublas/index.html#cublas-t-gemm
    // to understand the input parameters to cublasSgemm().
    for (int i = 0; i < num_reps; i++) {
        hipEventRecord(start);
        checkError(status);
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for SGEMM without Tensor Cores: %f ms\n", milliseconds);
    }

    // Copy the result back to the host.
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Check the result.
    #pragma omp parallel for
    for (int i = 0; i < n; i++) {
        #pragma omp parallel for
        for (int j = 0; j < n; j++) {
            float expected = 0.0f;
            #pragma omp parallel for reduction(+:expected)
            for (int k = 0; k < n; k++) {
                expected += h_A[k*n + j] * h_B[i*n + k];
            }
            // Check if the absolute difference of the cuBLAS output is within a small tolerance of the expected output.
            if (fabs(h_C[i*n + j] - expected) > 1e-1) {
               printf("Verification failed at index %d,%d! h_C[%d,%d] = %f, expected = %f\n", i, j, i, j, h_C[i*n + j], expected);
               exit(-1);
            }
        }
    }
    printf("Verification passed!\n\n");

    /****************************************************************************************************************************
                This next section is exactly the same as the last, however this time we pass cublasSetMathMode()
                CUBLAS_TF32_TENSOR_OP_MATH. This is the easiest way to enable acceleration of single-precision
                routines using TF32 tensor cores for those working on architectures Ampere and above.

                                    cublasSetMathMode(handle, CUBLAS_TF32_TENSOR_OP_MATH);

                                             NOTE: THIS WILL HAVE NO EFFECT ON VOLTA
     ***************************************************************************************************************************/

    status = hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    checkError(status);
    for (int i = 0; i < num_reps; i++) {
        hipEventRecord(start);
        checkError(status);
        status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Timie for SGEMM with Tensor Cores and TF32 (Ampere and beyond): %f ms\n", milliseconds);
    }

    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    #pragma omp parallel for
    for (int i = 0; i < n; i++) {
        #pragma omp parallel for
        for (int j = 0; j < n; j++) {
            float expected = 0.0f;
            #pragma omp parallel for reduction(+:expected)
            for (int k = 0; k < n; k++) {
                expected += h_A[k*n + j] * h_B[i*n + k];
            }
            if (fabs(h_C[i*n + j] - expected) > 1e-1) {
               printf("Verification failed at index %d,%d! h_C[%d,%d] = %f, expected = %f\n", i, j, i, j, h_C[i*n + j], expected);
               exit(-1);
            }
        }
    }
    printf("Verification passed!\n\n");


    /****************************************************************************************************************************
                This next section uses both tensor cores and mixed precision for acceleration on the volta architecture.
                To do this we tell cublasSetMathMode() to use the default math mode by passing CUBLAS_DEFAULT_MATH
                We then use the cublasSgemmEx() function to perform the sgemm. Note we explicitly tell it that our
                inputs are fp16.

                See here for further information:

                https://docs.nvidia.com/cuda/cublas/index.html#cublas-t-gemmex

     ***************************************************************************************************************************/

    // Allocate memory for fp16 arrays on the host
    half *h_hA = (half*)malloc(n * n * sizeof(half));
    half *h_hB = (half*)malloc(n * n * sizeof(half));

    // Allocate memory for fp16 on the device
    half *d_hA, *d_hB;

    hipMalloc((void**)&d_hA, n * n * sizeof(half));
    hipMalloc((void**)&d_hB, n * n * sizeof(half));

    // Initialise the matrices with data
    // that is suitable for the fp16 range
    for (int i = 0; i < n * n; i++) {
        float A = (float)(rand() % 100)/100.0f;
        float B = (float)(rand() % 100)/100.0f;
        h_hA[i] = (half)(A);
        h_hB[i] = (half)(B);
    }

    // Copy the data to the device
    hipMemcpy(d_hA, h_hA, n * n * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_hB, h_hB, n * n * sizeof(half), hipMemcpyHostToDevice);

    // Perform the matrix multiplication with Tensor Cores enabled and mixed precision
    status = hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    checkError(status);
    for (int i = 0; i < num_reps; i++) {
        hipEventRecord(start);
        status = cublasSgemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, d_hA, HIP_R_16F, n, d_hB, HIP_R_16F, n, &beta, d_C, HIP_R_32F, n);
        checkError(status);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Time for SGEMM with Tensor Cores and mixed precision (Volta): %f ms\n", milliseconds);
    }

    // Copy the result back to the host
    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    // Check the result
    #pragma omp parallel for
    for (int i = 0; i < n; i++) {
        #pragma omp parallel for
        for (int j = 0; j < n; j++) {
            float expected = 0.0f;
            #pragma omp parallel for reduction(+:expected)
            for (int k = 0; k < n; k++) {
                expected += (float)h_hA[k*n + j] * (float)h_hB[i*n + k];
            }
            //printf("\n%f\t%f", expected, h_C[i*n + j]);
            // Check if the absolute difference is within a small tolerance
            if (fabs(h_C[i*n + j] - expected) > 1e-1) {
               printf("Verification failed at index %d,%d! h_C[%d,%d] = %f, expected = %f\n", i, j, i, j, h_C[i*n + j], expected);
               exit(-1);
            }
        }
    }
    printf("Verification passed!\n\n");


    // Clean up
    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_hA);
    hipFree(d_hB);
    free(h_hA);
    free(h_hB);

    return 0;
}

